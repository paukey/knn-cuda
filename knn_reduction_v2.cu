
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<float.h> //DBL_MAX
#include <hip/hip_runtime_api.h>

#define restrict __restrict__
#define PADDINGCLASS -2
#define EXP 2
#define OUTPUT_FILE "ocuda"
#define INPUT_FILE "data"

void printStats(size_t bytes, hipEvent_t before, hipEvent_t after, const char *msg);
void check_error(hipError_t err, const char *msg);
void readInput(FILE* file, double* coords, double* coordsnew, int* classes, int spacedim, int numels, int newels);
void writeOutput(double* coords, int* classes, int spacedim, int numels);
__device__ int findMode(double4* elements, int classes_num, int k);
__global__ void findClass(double* coords, double* coordsnew, int* input_classes, double4* d_output, int spacedim, int classes_num, int numels, int offset, int newPointIndex, int newels, double* d_coordsDistances);
__device__ double distance(double* coords, double* coords2, int spacedim);
__global__ void findMin(double4* input, double* coords, double* coordsnew, int* classes, int classes_num, int spacedim, int numels, int offset, double4* result, int k, int newPointIndex, int eleInBlock, int newels, double* coordsDistances);
__device__ void swapdouble(double* x, double* y);
__device__ void swapInt(int* x, int* y);
__global__ void calcDistances(double* coords, double* coordsnew, int spacedim, int numels, int newels, double* coordsDistances);

//Declaration of shared-memory. It's going to contains partial minimum of distances
extern __shared__ double4 mPartial[];

int main(int argc, char *argv[])
{  
  int newels;                      //number of points we want classify
  int k;                           //number of nearest points we use to classify
  int numels;                      //total element already classified
  int spacedim;
  char filePath[255];              //path + filname of input file
  int classes_num;                 //number of classes
  double* h_coords;                //coords of existing points with a class
  double* h_coordsnew;             //coords of points we want to classify
  int* h_classes;                  //array contains the class for each points
  
  //*** Device-variables-declaration ***
  double* d_coords;
  double* d_coordsnew;
  double* d_coordsDistances;
  double* d_newcoordsDistances;
  double4* d_result; 
  int* d_classes;
  double4* d_output;
  //*** end-device-declaration
  
  //***cudaEvent-declaration***
  hipEvent_t before_allocation, before_input, before_upload, before_knn, before_download;
  hipEvent_t after_allocation, after_input, after_upload, after_knn, after_download;
  //***end-cudaEvent-declaration***
  
  if (argc > 2) 
  {
    strcpy(filePath, argv[1]);
    k = atoi(argv[2]);
  }
  else 
  {
    printf("how-to-use: knn <inputfile> <k> \n");
    exit(1);
  } 
  
  //***cuda-init-event***
  check_error(hipEventCreate(&before_allocation), "create before_allocation cudaEvent");
  check_error(hipEventCreate(&before_input), "create before_input cudaEvent");
  check_error(hipEventCreate(&before_upload), "create before_upload cudaEvent");
  check_error(hipEventCreate(&before_knn), "create before_knn cudaEvent");
  check_error(hipEventCreate(&before_download), "create before_download cudaEvent");
  
  check_error(hipEventCreate(&after_allocation), "create after_allocation cudaEvent");
  check_error(hipEventCreate(&after_input), "create after_input cudaEvent");
  check_error(hipEventCreate(&after_upload), "create after_upload cudaEvent");
  check_error(hipEventCreate(&after_knn), "create after_knn cudaEvent");
  check_error(hipEventCreate(&after_download), "create after_download cudaEvent");
  //***end-cuda-init-event***

  FILE *fp;
  if((fp = fopen(filePath, "r")) == NULL)
  {
        printf("No such file\n");
        exit(1);
  }
  
  fseek(fp, 0L, SEEK_END);
  float fileSize = ftell(fp);
  rewind(fp);
  
  int count = fscanf(fp, "%d,%d,%d,%d\n", &numels, &newels, &classes_num, &spacedim);
  int totalElements = numels + newels;

  //*** allocation ***
  hipEventRecord(before_allocation);
  h_coords = (double*) malloc(sizeof(double)*totalElements*spacedim);
  h_coordsnew = (double*) malloc(sizeof(double)*newels*spacedim);    
  h_classes = (int*) malloc(sizeof(int)*totalElements);
  
  const int blockSize = 512;
  int numBlocks = (totalElements + blockSize - 1)/blockSize;
  
  //*** device-allocation ***
  check_error(hipMalloc(&d_coords, totalElements*spacedim*sizeof(double)), "alloc d_coords_x");
  check_error(hipMalloc(&d_output, ((totalElements + blockSize - 1)/blockSize)*4*sizeof(double)), "alloc d_output");
  check_error(hipMalloc(&d_classes, totalElements*sizeof(int)), "alloc d_classes");
  check_error(hipMalloc(&d_result, 4*k*sizeof(double)), "alloc d_result");
  check_error(hipMalloc(&d_coordsDistances, (newels*totalElements)*sizeof(double)), "alloc d_coordsDistances");
  check_error(hipMalloc(&d_newcoordsDistances, (newels*newels)*sizeof(double)), "alloc d_newcoordsDistances");
  check_error(hipMalloc(&d_coordsnew, newels*spacedim*sizeof(double)), "alloc d_coordsnew");
  //*** end-device-allocation ***
  hipEventRecord(after_allocation);
  
  ///***input-from-file***
  hipEventRecord(before_input);
  readInput(fp, h_coords, h_coordsnew, h_classes, spacedim, numels, newels);
  hipEventRecord(after_input);
  fclose(fp);
  ///***end-input-from-file***

  //***copy-arrays-on-device***
  hipEventRecord(before_upload);
  check_error(hipMemcpy(d_coords, h_coords, totalElements*spacedim*sizeof(double), hipMemcpyHostToDevice), "copy d_coords");
  check_error(hipMemcpy(d_classes, h_classes, totalElements*sizeof(int), hipMemcpyHostToDevice), "copy d_classes");
  check_error(hipMemcpy(d_coordsnew, h_coordsnew, newels*spacedim*sizeof(double), hipMemcpyHostToDevice), "copy d_coordsnew");
  hipEventRecord(after_upload);
  //***end-copy-arrays-on-device***
  
    hipEventRecord(before_knn);
  calcDistances<<<numBlocks, blockSize>>>(d_coords, d_coordsnew, spacedim, numels, newels, d_coordsDistances);
  

  int i, j;
  for (i = 0; i < newels; i++)
  {
    numBlocks = (numels + blockSize - 1)/blockSize;
    j = 0;
    for (j = 0; j < k; j++)
    {
      findClass<<<numBlocks, blockSize, blockSize*4*sizeof(double)>>>(
      d_coords, d_coordsnew, d_classes,
      d_output,
      spacedim, classes_num,
      numels, j, i, newels, d_coordsDistances);
      
      findMin<<<1, blockSize, blockSize*4*sizeof(double)>>>(d_output, d_coords, d_coordsnew, d_classes, classes_num, spacedim, numels, j, d_result, k, i, numBlocks, newels, d_coordsDistances);
    }
    numels++;
  }
  hipEventRecord(after_knn);
  
  hipEventRecord(before_download);
  check_error(hipMemcpy(h_coords, d_coords, spacedim*totalElements*sizeof(double), hipMemcpyDeviceToHost), "download coords");
  check_error(hipMemcpy(h_classes, d_classes, totalElements*sizeof(int), hipMemcpyDeviceToHost), "download classes");
  hipEventRecord(after_download);
  
  check_error(hipEventSynchronize(after_download), "sync cudaEvents");
  printStats((totalElements+newels)*(1+spacedim)*sizeof(double) + totalElements*sizeof(int), before_allocation, after_allocation, "[time] allocation");
  printStats(fileSize, before_input, after_input, "[time] read input file");
  printStats(fileSize, before_upload, after_upload, "[time] upload host->device");
  printStats((spacedim*totalElements*sizeof(double) + totalElements*sizeof(int))*newels, before_knn, after_knn, "[time] knn algorithm");
  printStats((spacedim*totalElements*sizeof(double) + totalElements*sizeof(int))*newels, before_download, after_download, "[time] download device->host");
  
  writeOutput(h_coords, h_classes, spacedim, numels);
  return 0;
}

void check_error(hipError_t err, const char *msg)
{
  if (err != hipSuccess) 
  {
    fprintf(stderr, "%s : error %d (%s)\n", msg, err, hipGetErrorString(err));
    exit(err);
  }
}

float runtime;
void printStats(size_t bytes, hipEvent_t before, hipEvent_t after, const char *msg)
{ 
  check_error(hipEventElapsedTime(&runtime, before, after), msg);
  printf("%s %gms, %g GB/s\n", msg, runtime, bytes/runtime/(1024*1024));
}

//Parallel reduction to find the k-minimum distances
__global__ void findClass(
  double* coords, double* coordsnew,
  int* input_classes, double4* d_output,
  int spacedim, int classes_num, int numels, int offset, int newPointIndex, int newels, double* d_coordsDistances)
{
  int gid = offset + threadIdx.x + blockIdx.x*blockDim.x;
  int lid = threadIdx.x;
  mPartial[lid] = make_double4(-1, PADDINGCLASS, -1, -1);
  if (gid >= numels) return;  
  
  double min = d_coordsDistances[gid*newels + newPointIndex];
  double d;
  int c = input_classes[gid];
  int minID = gid;

  while (gid < numels)
  {
    d = d_coordsDistances[gid*newels + newPointIndex];
    if(d < min)
    {
      min = d;
      minID = gid;
      c = input_classes[gid];
    }
    gid += gridDim.x*blockDim.x;
  }

  mPartial[lid] = make_double4(min, (double)c, minID, -1);
  
  //Part 2: reduction in shared memory
  int stride = (blockDim.x)/2;
  while (stride > 0)
  {
    __syncthreads();
    if (lid < stride && mPartial[lid+stride].y != PADDINGCLASS && mPartial[lid].y != PADDINGCLASS && mPartial[lid+stride].x < mPartial[lid].x)
        mPartial[lid] = mPartial[lid+stride];
    stride /= 2;
  }

  /* Part 3: save the block's result in global memory */
  if (lid == 0)
    d_output[blockIdx.x] = mPartial[0];
}

__global__ void findMin(double4* input, double* coords, double* coordsnew, int* classes, int classes_num, int spacedim, int numels, int offset, double4* result, int k, int newPointIndex, int eleInBlock, int newels, double* coordsDistances)
{
  int gid = threadIdx.x + blockIdx.x*blockDim.x;
  int lid = threadIdx.x;
  mPartial[lid] = make_double4(-1, PADDINGCLASS, -1, -1);
  if (gid >= eleInBlock || gid >= blockDim.x) return;

  double distmin = input[gid].x;
  double classmin = input[gid].y;
  double gidMin = input[gid].z;
       
  while (gid < eleInBlock)
  {
    if(input[gid].x < distmin)
    {
      distmin = input[gid].x;
      classmin = input[gid].y;
      gidMin = input[gid].z;
    }
    gid += gridDim.x*blockDim.x;
  }
    
  mPartial[lid] = make_double4(distmin, classmin, gidMin, -1);

  //Part 2: reduction in shared memory
  int stride = (blockDim.x)/2;
  while (stride > 0)
  {
    __syncthreads();
    if (lid < stride && mPartial[lid+stride].y != PADDINGCLASS && mPartial[lid].y != PADDINGCLASS && mPartial[lid+stride].x < mPartial[lid].x)
      mPartial[lid] = mPartial[lid + stride];
    stride /= 2;
  }

  /* Part 3: save the block's result in global memory */
  if (lid == 0)
  {    
    input[0] = mPartial[0];
    int minID = mPartial[0].z;
    
    int i = 0;
    for (i = 0; i < spacedim; i++)
      swapdouble(&(coords[spacedim*minID+i]), &(coords[offset*spacedim+i]));
    
    for (i = 0; i < newels; i++)
      swapdouble(&(coordsDistances[newels*minID + i]), &(coordsDistances[newels*offset+i]));
      
    swapInt(&(classes[minID]), &(classes[offset]));    
    result[offset] = input[0];
    if (offset == k-1)
    {
      int j;
      for (j = 0; j < spacedim; j++)
          coords[spacedim*numels+j] = coordsnew[spacedim*newPointIndex + j];
                
      classes[numels] = findMode(result, classes_num, k);
    }
  }
}

__global__ void calcDistances(double* coords, double* coordsnew, int spacedim, int numels, int newels, double* coordsDistances)
{
  int point = threadIdx.x + blockIdx.x*blockDim.x;
  int totalElements = numels + newels;
  if (point >= totalElements) return;
  
  int i = 0;
  if (point < numels)
  {
    //per ogni punto - mi calcolo le distanze con i punti newles
    for (i = 0; i < newels; i++)
      coordsDistances[point*newels+i] = distance((point*spacedim+coords), (i*spacedim+coordsnew), spacedim);  
  }
  else 
  {
    //punto da determinare, mi calcolo la distanza con il resto dei punti newels
    int index = point - numels;
    for (i = 0; i < newels; i++)
    {
        //distance per me stesso
        if (i == index)
          coordsDistances[point*newels+i] = DBL_MAX;
        else
          coordsDistances[point*newels+i] = distance((index*spacedim+coordsnew), (i*spacedim+coordsnew), spacedim);
    }
  }  
}

// read input from file
void readInput(FILE* file, double* coords, double* coordsnew, int* classes, int spacedim, int numels, int newels)
{
  int i, j;
  int count;
  for(i=0; i<numels; i++)
  {
    for (j = 0; j < spacedim; j++)
      count = fscanf(file, "%lf,", &(coords[i*spacedim +j]));
    count = fscanf(file, "%d\n", &(classes[i]));
  }
   
  for(i = 0; i < newels; i++)
  {
    for (j = 0; j < spacedim; j++)
      count = fscanf(file, "%lf,", &(coordsnew[i*spacedim+j]));
    count = fscanf(file, "-1\n");
  }
  count++;
}

//Write Output on file
void writeOutput(double* coords, int* classes, int spacedim, int numels)
{
  FILE *fp;
  fp = fopen(OUTPUT_FILE, "w");
  int i, j;
  for( i = 0; i < numels; i++)
  {
    for (j = 0; j < spacedim; j++)
      fprintf(fp, "%lf,", coords[i*spacedim+j]);
    
    fprintf(fp, "%d\n", classes[i]);
  }
  fclose(fp); 
}

//multidimensional euclidian distance
__device__ double distance(double* coords, double* coords2, int spacedim)
{
  double sum = 0;
  int i;
  for (i = 0; i < spacedim; i++)
  {
    double diff = coords[i] - coords2[i];
    sum += diff*diff;
  }  
  return sum;
}

__device__ void swapdouble(double* x, double* y)
{
  double tmp = *x;
  *x = *y;
  *y = tmp;
}

__device__ void swapInt(int* x, int* y)
{
  int tmp = *x;
  *x = *y;
  *y = tmp;
}

__device__ int findMode(double4* elements, int classes_num, int k)
{
  int* classCount = (int*) (malloc(sizeof(int)*classes_num));
  int i;
  for (i = 0; i < classes_num; i++)
    classCount[i] = 0;
       
  for (i = 0; i < k; i++)
    classCount[(int)(elements[i].y)]++;
    
  int max = 0;
  int maxValue = classCount[0];
  for (i = 1; i < classes_num; i++)
  {
    int value = classCount[i];
    if (value > maxValue)
    {
      max = i;
      maxValue = value;
    }
    else if (value != 0 && maxValue == value)
    {
        int j = 0;
        for (j = 0; j < k; j++)
        {
          if (elements[j].y == i)
          {
            max = i;
            break;
          }
          else if (elements[j].y == max)
            break;
        }
    }
  }
  
  free(classCount);
  return max;
}
